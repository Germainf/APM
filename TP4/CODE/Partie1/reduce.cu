
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

__global__ void reduce_kernel(float *in, float *out, int ntot)
{
	int i=blockDim.x*blockIdx.x + threadIdx.x;

	for(int j=1 ; j<blockDim.x ; j*=2)
	{
		if(!(threadIdx.x & (2*j-1)) && i+j<ntot)
			in[i]+=in[i+j];
		__syncthreads();
	}

	if(threadIdx.x == 0)
		out[blockIdx.x]=in[i];
}

__host__ void init_vec(float *h_in, int ntot)
{
    for(int i = 0 ; i < ntot ; i++)
    {
	h_in[i] = sinf(float(i));
    }
}

__host__ void verif(float sum, float *h_in, int ntot)
{
    float sum_res = 0.;
    for(int i = 0 ; i < ntot ; i++)
    {
	sum_res += h_in[i];
    }
    float err = fabsf((sum - sum_res)/sum);
    printf("GPU sum : %.4e\n", sum);
    printf("CPU sum : %.4e\n", sum_res);
    if (err < 1.e-4)
    {
	printf("TEST PASSED (err %.4e < 1.e-4).\n", err);
    }
    else
    {
	printf("TEST FAILED (err %.4e > 1.e-4).\n", err);
    }
}

int main(int argc, char **argv)
{
    float sum;
    int nthreads, nblocks, ntot;

    nthreads = 128;
    ntot = atoi(argv[1]);
    nblocks = (ntot + nthreads - 1) / nthreads;

    printf("Ntot     : %d\n", ntot);
    printf("nthreads : %d\n", nthreads);
    printf("nblocks  : %d\n", nblocks);

    float *d_sum, *d_bl, *d_in, *h_in;

    h_in = (float*)malloc(ntot*sizeof(float));

    hipMalloc((void**)&d_sum, sizeof(float));
    hipMalloc((void**)&d_bl, nblocks*sizeof(float));
    hipMalloc((void**)&d_in, ntot*sizeof(float));

    init_vec(h_in, ntot);
    hipMemcpy(d_in, h_in, ntot*sizeof(float), hipMemcpyHostToDevice);

    // TODO : la réduction de d_in a lieu ici, le resultat est obtenu dans *d_sum
    reduce_kernel<<<nblocks, nthreads>>>(d_in, d_bl, ntot);
    reduce_kernel<<<1, nblocks>>>(d_bl, d_sum, nblocks);

    hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
    
    verif(sum, h_in, ntot);

    hipFree(d_sum);
    hipFree(d_bl);
    hipFree(d_in);
    free(h_in);

    return 0;
}

